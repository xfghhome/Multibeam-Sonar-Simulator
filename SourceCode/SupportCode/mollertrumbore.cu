#include "hip/hip_runtime.h"

/*
 * Fast detection test if a ray intersects with a set of triangles using CUDA accelerated kernels based on the work of Möller and Trumbore (1997).
 *
 * Arguments: 'vertices' [(3 * 3 * numTriangles),type float32]
                These are interleaved, example:                 
                vertices=[T0_V0_X, T0_V0_Y, T0_V0_Z;
                         T0_V1_X, T0_V1_Y, T0_V1_Z;
                         T0_V2_X, T0_V2_Y, T0_V2_Z;
                         T1_V0_X, T1_V0_Y, T1_V0_Z;
                         T1_V1_X, T1_V1_Y, T1_V1_Z;
                         T1_V2_X, T1_V2_Y, T1_V2_Z;
                         ...]
                 with Ti: triangles, Vj:vertices 1 to 3 of triangle and X/Y/Z the 3 coordinates
              'raysFrom' [(3 * numRays),type float32]
                These are the starting points of each ray in X Y Z
              'directions' [(3 * numRays),type float32]
                These are the directions vectors of each ray in X Y Z
              
 * Returns:   'results' [numTriangles * numRays, type int32]
                These are the binary test results to see if there is an intersection.
                1 = intersection found, 0 = no intersection
                
 * Compile with 'mexcuda -v mollertrumbore.cu' (-v for extra details for debugging)
 * Requires CUDA toolkit and a C compiler.
 * Make sure to correctly set c++ compiler with 'mex -setup c++' and clicking on the link of the version you want if asked.
 * And make sure to set the CUDA enviroment variable correctly with
 * 'setenv('MW_NVCC_PATH','/usr/local/cuda-X/bin')' on Linux
 * 'setenv('MW_NVCC_PATH','C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\vX\bin')' on Windows
 * 
 * Original CUDA implementation of Möller and Trumbore is by Raymond Leung (2022), 
 * 'GPU implementation of a ray-surface intersection algorithm in CUDA',
 * arXiv e-print 2209.02878, 2022.
 * Source code available at: https://github.com/raymondleung8/gpu-ray-surface-intersection-in-cuda
 * and is made under a BSD 3 license.
 *
 * MATLAB implementation by Wouter Jansen, Cosys-Lab, University of Antwerp
 */

#define EPSILON 0.0000000001
#include <string>   

__device__ void subtract(const float *a, const float *b, float *out)
{
    out[0] = a[0] - b[0];
    out[1] = a[1] - b[1];
    out[2] = a[2] - b[2];
}

__device__ void dot(const float *a, const float *b, float &out)
{
    out = a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
}

__device__ void cross(const float *a, const float *b, float *out)
{
    out[0] = a[1]*b[2] - a[2]*b[1];
    out[1] = a[2]*b[0] - a[0]*b[2];
    out[2] = a[0]*b[1] - a[1]*b[0];
}

__device__ float tolerance(const float *d, const float *eAB, const float *eAC)
{
    float scaling = sqrtf((d[0]*d[0] + d[1]*d[1] + d[2]*d[2]) *
                          (eAB[0]*eAB[0] + eAB[1]*eAB[1] + eAB[2]*eAB[2]) *
                          (eAC[0]*eAC[0] + eAC[1]*eAC[1] + eAC[2]*eAC[2]));
    return (scaling > 1)? scaling * EPSILON : EPSILON;
}

// Implement the Moller-Trumbore ray-triangle intersection algorithm
// - Ray model: R(t) = Q0 + t *(dir), where Q0 denote segment start point
// - and dir the direction of the segment
// - Point on triangle: T(u,v) = (1-u-v)*V0 + u*V1 + v*V2
//
__device__ int intersectMoller(
                const float *v0, const float *v1, const float *v2,
                const float *edge1, const float *edge2,
                const float *q0, const float *dir)
{
    float avec[3], bvec[3], tvec[3], t, u, v, det, inv_det;
    cross(dir, edge2, avec);
    dot(avec, edge1, det);
    float epsilon = tolerance(dir, edge1, edge2);
    if (det > epsilon) {
        subtract(q0, v0, tvec);
        dot(avec, tvec, u);
        if (u < 0 || u > det){
            return 0;
        }
        cross(tvec, edge1, bvec);
        dot(bvec, dir, v);
        if (v < 0 || u + v > det){
            return 0;
        }
    }
    else if (det < -epsilon) {
        subtract(q0, v0, tvec);
        dot(avec, tvec, u);
        if (u > 0 || u < det){
            return 0;
        }
        cross(tvec, edge1, bvec);
        dot(bvec, dir, v);
        if (v > 0 || u + v < det){
            return 0;
        }
    }
    else{
        return 0;
    }
    inv_det = 1.0 / det;
    dot(bvec, edge2, t);
    t *= inv_det;
    if (t < 0) {
        return 0;
    }
    else {
        return 1;
    }
}

__device__ void checkRayTriangleIntersection(const float* __restrict__ vertices,
                                             const float* __restrict__ raysFrom,
                                             const float* __restrict__ directions,
                                             int* __restrict__ results,
                                             int triangleIdx, int rayIdx,
                                             int numTriangles, int numRays)
{
    float triangleVerts[9], edge1[3], edge2[3];
    const float *v0 = &triangleVerts[0],
                *v1 = &triangleVerts[3],
                *v2 = &triangleVerts[6];
    for(int j = 0; j < 3; j++) { // loop by triangle v0, v1, v2
        for (int k = 0; k < 3; k++) { // loop by x, y and z
            triangleVerts[3*j + k] = vertices[9*(triangleIdx) + (3*j) + k];
        }
    }
    subtract(v1, v0, edge1);
    subtract(v2, v0, edge2);
   
    // Apply Moller-Trumbore ray-triangle intersection test
    const float *start = &raysFrom[3*rayIdx], *dir = &directions[3*rayIdx];
    if (intersectMoller(v0, v1, v2, edge1, edge2, start, dir)) {
        // printf("HIT! triangleIdx: %i rayIdx: %i\n", triangleIdx, rayIdx);
        int tempRayIdx = rayIdx / 32;
        int bitOffset = rayIdx % 32;
        atomicOr(&results[tempRayIdx * numTriangles + triangleIdx], 1 << bitOffset);
    }
}

__global__ void checkRayTriangleIntersectionKernel(const float* vertices,
                                                   const float* raysFrom,
                                                   const float* directions,
                                                   int* results,
                                                   int numTriangles, int numRays)
{
    int triangleIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int rayIdx = blockIdx.y * blockDim.y + threadIdx.y;
    if(triangleIdx < numTriangles && rayIdx < numRays){
        checkRayTriangleIntersection(vertices, raysFrom, directions, results, triangleIdx, rayIdx, numTriangles, numRays);
    }
}

#include "mex.h"
#include "gpu/mxGPUArray.h"   

/*
 * Host code for CPU
 */
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
    /* Declare all variables.*/
    mxGPUArray const *deviceVertices;
    mxGPUArray const *deviceRaysFrom;
    mxGPUArray const *deviceDirections;
    mxGPUArray *deviceResults;
    float const *d_vertices;
    float const *d_raysFrom;
    float const *d_directions;
    int *d_results;
    int numTriangles;
    int numRays;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Throw an error if the input are not a CPU arrays. */
    if ((mxIsGPUArray(prhs[0])) || (mxIsGPUArray(prhs[1])) || (mxIsGPUArray(prhs[2]))) {
        mexErrMsgIdAndTxt("parallel:gpu:mexGPUExample:InvalidInput", "The input matrices have to be normal CPU arrays, not GPUArrays.\n");
    }

    /* Throw an error if the input are not the correct datatype. */
    if ( mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt("parallel:gpu:mexGPUExample:InvalidInput", "The vertices data matrix has to be of datatype 'single'.\n");
    }
    if ( mxGetClassID(prhs[1]) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt("parallel:gpu:mexGPUExample:InvalidInput", "The ray-from data matrix has to be of datatype 'single'.\n");
    }
    if ( mxGetClassID(prhs[2]) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt("parallel:gpu:mexGPUExample:InvalidInput", "The ray-to data matrix has to be of datatype 'single'.\n");
    }

    deviceVertices = mxGPUCreateFromMxArray(prhs[0]);
    deviceRaysFrom = mxGPUCreateFromMxArray(prhs[1]);
    deviceDirections = mxGPUCreateFromMxArray(prhs[2]);
    numTriangles = (int)((float)mxGPUGetDimensions(deviceVertices)[1]) / 3;
    numRays = mxGPUGetDimensions(deviceRaysFrom)[1];

    /* Extract a pointer to the input data on the device. */
    d_vertices = (float const *)(mxGPUGetDataReadOnly(deviceVertices));
    d_raysFrom = (float const *)(mxGPUGetDataReadOnly(deviceRaysFrom));
    d_directions = (float const *)(mxGPUGetDataReadOnly(deviceDirections));

    // printf("numTriangles:%i numRays:%i \n", numTriangles, numRays);

    /* Create a GPUArray to hold the result and get its underlying pointer. */
    int numRaysLogicalStorage = ceil((float)numRays / (float)32);
    mwSize dims[2] = {numTriangles, numRaysLogicalStorage};
    // printf("Storing results in array of %ix%i\n", numTriangles, numRaysLogicalStorage);
    deviceResults = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(deviceVertices),
                                        dims,
                                        mxINT32_CLASS,
                                        mxREAL,
                                        MX_GPU_INITIALIZE_VALUES);
    d_results = (int *)(mxGPUGetData(deviceResults));

    /* Execute the kernel. */
    dim3 threadsPerBlock(256, 4, 1);
    int gridX = (int)ceil(numTriangles / (threadsPerBlock.x*1.0));
    int gridY = (int)ceil(numRays / (threadsPerBlock.y*1.0));
    dim3 numBlocks(gridX, gridY, 1);
    checkRayTriangleIntersectionKernel<<<numBlocks, threadsPerBlock>>>(d_vertices, d_raysFrom, d_directions, d_results, numTriangles, numRays);

    /* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnCPU(deviceResults);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the mex function.
     */
    mxGPUDestroyGPUArray(deviceVertices);
    mxGPUDestroyGPUArray(deviceRaysFrom);
    mxGPUDestroyGPUArray(deviceDirections);
    mxGPUDestroyGPUArray(deviceResults);
}
